/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__global__ void gpuYoloLayer(
    const float* input, int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, int* countData,
    const float scoreThreshold, const uint netWidth, const uint netHeight, const uint gridSizeX, const uint gridSizeY,
    const uint numOutputClasses, const uint numBBoxes, const float scaleXY, const float* anchors, const int* mask)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
        return;

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float objectness
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    if (objectness < scoreThreshold)
        return;

    int count = (int)atomicAdd(&countData[0], 1);

    const float alpha = scaleXY;
    const float beta = -0.5 * (scaleXY - 1);

    float x
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)])
          * alpha + beta + x_id) * netWidth / gridSizeX;

    float y
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)])
          * alpha + beta + y_id) * netHeight / gridSizeY;

    float w
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)])
          * anchors[mask[z_id] * 2];

    float h
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)])
          * anchors[mask[z_id] * 2 + 1];

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    d_indexes[count] = count;
    d_scores[count] = objectness * maxProb + 1.f;
    d_boxes[count * 4 + 0] = x - 0.5 * w;
    d_boxes[count * 4 + 1] = y - 0.5 * h;
    d_boxes[count * 4 + 2] = x + 0.5 * w;
    d_boxes[count * 4 + 3] = y + 0.5 * h;
    d_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer(
    const void* input, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer(
    const void* input, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * inputSize),
            reinterpret_cast<int*>(d_indexes) + (batch * outputSize),
            reinterpret_cast<float*>(d_scores) + (batch * outputSize),
            reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize), reinterpret_cast<int*>(countData) + (batch),
            scoreThreshold, netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, scaleXY,
            reinterpret_cast<const float*>(anchors), reinterpret_cast<const int*>(mask));
    }
    return hipGetLastError();
}
