/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void softmaxGPU(
    const float* input, const int bbindex, const int numGridCells, uint z_id, const uint numOutputClasses, float temp,
    float* output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < numOutputClasses; ++i) {
        int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
        largest = (val>largest) ? val : largest;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        float e = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
        sum += e;
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
    }
}

__global__ void gpuRegionLayer(
    const float* input, float* softmax, int* d_indexes, float* d_scores, float* d_boxes, int* d_classes, int* countData,
    const float scoreThreshold, const uint netWidth, const uint netHeight, const uint gridSizeX, const uint gridSizeY,
    const uint numOutputClasses, const uint numBBoxes, const float* anchors)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
        return;

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float objectness
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    if (objectness < scoreThreshold)
        return;

    int count = (int)atomicAdd(&countData[0], 1);

    float x
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)])
          + x_id) * netWidth / gridSizeX;

    float y
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)])
          + y_id) * netHeight / gridSizeY;

    float w
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)])
          * anchors[z_id * 2] * netWidth / gridSizeX;

    float h
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)])
          * anchors[z_id * 2 + 1] * netHeight / gridSizeY;

    softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0, softmax);

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = softmax[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    d_indexes[count] = count;
    d_scores[count] = objectness * maxProb + 1.f;
    d_boxes[count * 4 + 0] = x - 0.5 * w;
    d_boxes[count * 4 + 1] = y - 0.5 * h;
    d_boxes[count * 4 + 2] = x + 0.5 * w;
    d_boxes[count * 4 + 3] = y + 0.5 * h;
    d_classes[count] = maxIndex;
}

hipError_t cudaRegionLayer(
    const void* input, void* softmax, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const void* anchors, hipStream_t stream);

hipError_t cudaRegionLayer(
    const void* input, void* softmax, void* d_indexes, void* d_scores, void* d_boxes, void* d_classes, void* countData,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const void* anchors, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * inputSize),
            reinterpret_cast<float*>(softmax) + (batch * inputSize),
            reinterpret_cast<int*>(d_indexes) + (batch * outputSize),
            reinterpret_cast<float*>(d_scores) + (batch * outputSize),
            reinterpret_cast<float*>(d_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(d_classes) + (batch * outputSize), reinterpret_cast<int*>(countData) + (batch),
            scoreThreshold, netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
            reinterpret_cast<const float*>(anchors));
    }
    return hipGetLastError();
}
